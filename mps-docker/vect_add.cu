#include <hip/hip_runtime.h>
#include <iostream>


const int N = 100000000;

__global__ void memoryIntensiveKernel(float *a, float *b, float *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
  	{
    	c[i] = a[i] + b[i];
		  b[i] = a[i] + c[i];
		  a[i] = b[i] + c[i];
  	}
  
}

int main()
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *a, *b, *c;
  hipMalloc((void **)&a, N * sizeof(float));
  hipMalloc((void **)&b, N * sizeof(float));
  hipMalloc((void **)&c, N * sizeof(float));

  float *a_h, *b_h;
  a_h = new float[N];
  b_h = new float[N];

  for (int i = 0; i < N; i++)
  {
    a_h[i] = i;
    b_h[i] = i;
  }

  hipMemcpy(a, a_h, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b, b_h, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 512;
  int numBlocks = (N + blockSize - 1) / blockSize;
  hipEventRecord(start);
  for (int iter=1; iter<5001; iter++)
  {  
  	memoryIntensiveKernel<<<numBlocks, blockSize>>>(a, b, c);
	  std::cout << iter <<" Iteration completed"<<std::endl;
  }
  hipEventRecord(stop);

  float *c_h;
  c_h = new float[N];
  hipMemcpy(c_h,c,N*sizeof(float),hipMemcpyDeviceToHost);

  hipFree(a);
  hipFree(b);
  hipFree(c);

  hipEventSynchronize(stop);
  float mil = 0;
  hipEventElapsedTime(&mil,start,stop);
  printf("ET: %f ms\n",mil);
  return 0;
}
